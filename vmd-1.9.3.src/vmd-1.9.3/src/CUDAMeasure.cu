/***************************************************************************
 *cr
 *cr            (C) Copyright 2007-2012 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/***************************************************************************
 * RCS INFORMATION:
 *
 *      $RCSfile: CUDAMeasureRDF.cu,v $
 *      $Author: johns $        $Locker:  $             $State: Exp $
 *      $Revision: 1.25 $      $Date: 2011/01/18 15:32:28 $
 *
 ***************************************************************************
 * DESCRIPTION:
 *   CUDA accelerated analysis routines
 *
 ***************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "Inform.h"
#include "utilities.h"
#include "WKFThreads.h"
#include "WKFUtils.h"
#include "CUDAKernels.h" 

